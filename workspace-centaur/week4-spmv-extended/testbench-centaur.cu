#include "hip/hip_runtime.h"
// Week 4
// cuSPARSE vs Kernel Testbench.
// acoustikue@yonsei.ac.kr
// written by SukJoon Oh

#include <hip/hip_runtime.h>
// #include <>
#include <hipsparse.h>	// hipsparseSpMV

#include <stdio.h>
#include <stdlib.h>

#include "mmio.h"
#include "errchk.cuh"
#include "spmv_kernel.cuh"

// Option settings

// #define COO
#define CSR

// #define CUSPARSE
#define SCALAR_KERNEL
// #define VECTOR_KERNEL

// ---- main() ----
// Entry
int main(int argc, char* argv[])
{
	int test_iterations 	= 0;
	
	int N 					= 0;
	int M 					= 0;
	int NZ 					= 0;

	int* host_JR			= NULL;
	int* host_JC			= NULL;
    float* host_AA			= NULL;
	int* host_P				= NULL;

	int* device_JR			= NULL;
	int* device_JC			= NULL;
	float* device_AA		= NULL;
	float* device_AA_sorted	= NULL;
	int* device_P			= NULL;

	void* buffer			= NULL;
	size_t buffer_size		= 0;

	hipsparseHandle_t handle = NULL;
	hipStream_t stream		= NULL;

    if (argc == 1 || argc == 2) { printf("Too few arguments.\nProgram exit.\n"); exit(0); }
    if (argc >= 4) 				{ printf("Too many argmuments.\nProgram exit.\n"); exit(0); }

    test_iterations = atoi(argv[1]);
    printf("(arg1) Target iterations: %d\n", test_iterations);
	printf("(arg2) File name: %s\n", argv[2]);

	//
	// Reading file
	{
		FILE* MTX;
		MTX = fopen(argv[2], "r");
		
		MM_typecode matrix_code;
		
		printf("Reading %s... \n", argv[2]);

		// Read banner, type, etc essential infos
		// Verification steps are ignored.
		if (mm_read_banner(MTX, &matrix_code) != 0) exit(1);
		mm_read_mtx_crd_size(MTX, &M, &N, &NZ); // Over max 1025

		host_JR	    = (int*)malloc(NZ * sizeof(int));
		host_JC	    = (int*)malloc(NZ * sizeof(int));
		host_AA	    = (float*)malloc(NZ * sizeof(float));
		host_P	    = (int*)malloc(NZ * sizeof(int));

		for (register int i = 0; i < NZ; i++)
			fscanf(MTX, "%d %d %f\n", &host_JR[i], &host_JC[i], &host_AA[i]);

		fclose(MTX);
	}


	// ---- Step 1. Load info ----	
	printf("(File info)\tm : %d, n : %d, nz : %d\n", M, N, NZ);
	printf("Printing samples...\n");
	printf("JR: "); for (register int i = 0; i < 10; i++) printf("%6.0d", host_JR[i]); printf("\n");
	printf("JC: "); for (register int i = 0; i < 10; i++) printf("%6.0d", host_JC[i]); printf("\n");
	printf("AA: "); for (register int i = 0; i < 10; i++) printf("%6.0lf", host_AA[i]); printf("\n");
	printf("File successfully loaded.\n");

	// ---- Step 2. Handle create, bind a stream ---- 
	printf("Preparing for cusparseXcoosort...\n");
	CUDA_ERR(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	CUSPARSE_ERR(hipsparseCreate(&handle));
	CUSPARSE_ERR(hipsparseSetStream(handle, stream));

	// ---- Step 3. Allocate Buffer ---- 
	CUSPARSE_ERR(hipsparseXcoosort_bufferSizeExt(handle, M, N, NZ, device_JR, device_JC, &buffer_size));
	printf("Buffer allocation for hipsparseXcoosortByRow : %ld Byte\n", buffer_size);

	CUDA_ERR(hipMalloc((void**)&device_JR, sizeof(int) * NZ));
	CUDA_ERR(hipMalloc((void**)&device_JC, sizeof(int) * NZ));
	CUDA_ERR(hipMalloc((void**)&device_P, sizeof(int) * NZ));
	CUDA_ERR(hipMalloc((void**)&device_AA, sizeof(float) * NZ));
	CUDA_ERR(hipMalloc((void**)&device_AA_sorted, sizeof(float) * NZ));
	CUDA_ERR(hipMalloc((void**)&buffer, sizeof(char) * buffer_size));

	CUDA_ERR(hipMemcpy(device_JR, host_JR, sizeof(int) * NZ, hipMemcpyHostToDevice));
	CUDA_ERR(hipMemcpy(device_JC, host_JC, sizeof(int) * NZ, hipMemcpyHostToDevice));
	CUDA_ERR(hipMemcpy(device_AA, host_AA, sizeof(float) * NZ, hipMemcpyHostToDevice));
	CUDA_ERR(hipDeviceSynchronize());

	printf("Allocation/Memcopy to GPU done.\n");

	// ---- Step 4. Setup permutation vector P to Identity ---- 
	CUSPARSE_ERR(hipsparseCreateIdentityPermutation(handle, NZ, device_P));

	// ---- Step 5. Sort ---- 
	CUSPARSE_ERR(hipsparseXcoosortByRow(handle, M, N, NZ, device_JR, device_JC, device_P, buffer));
	printf("hipsparseXcoosortByRow done.\n");

	// Gather
	CUSPARSE_ERR(hipsparseSgthr(handle, NZ, device_AA, device_AA_sorted, device_P, HIPSPARSE_INDEX_BASE_ZERO));
	CUDA_ERR(hipDeviceSynchronize());

	// Fetch back
	CUDA_ERR(hipMemcpy(host_JR, device_JR, sizeof(int) * NZ, hipMemcpyDeviceToHost));
	CUDA_ERR(hipMemcpy(host_JC, device_JC, sizeof(int) * NZ, hipMemcpyDeviceToHost));
	CUDA_ERR(hipMemcpy(host_P, device_P, sizeof(int) * NZ, hipMemcpyDeviceToHost));
	CUDA_ERR(hipMemcpy(host_AA, device_AA_sorted, sizeof(float) * NZ, hipMemcpyDeviceToHost));
    CUDA_ERR(hipDeviceSynchronize());
    
    // Free memories
    if (device_P) 	hipFree(device_P);
	if (device_AA) 	hipFree(device_AA);
	if (buffer) 	hipFree(buffer);
	if (handle) 	hipsparseDestroy(handle);
	if (stream) 	hipStreamDestroy(stream);

	free(host_P); // Unnecessary

	printf("Printing sorted values...\n");
	printf("JR: "); for (register int i = 0; i < 10; i++) printf("%6.0d", host_JR[i]); printf("\n");
	printf("JC: "); for (register int i = 0; i < 10; i++) printf("%6.0d", host_JC[i]); printf("\n");
	printf("AA: "); for (register int i = 0; i < 10; i++) printf("%6.0lf", host_AA[i]); printf("\n");

#ifdef CSR
	printf("Converting COO to CSR...\n");

	if (device_JR) 	hipFree(device_JR);
	if (device_JC) 	hipFree(device_JC);

    int* t_JR	    = (int*)calloc((M + 1), sizeof(int));
	for (int i = 0; i < M + 1; i++) t_JR[i]++; 
	for (int i = 0; i < NZ; i++) 	t_JR[host_JR[i]]++;
	for (int i = 0; i < M; i++)		t_JR[i + 1] += (t_JR[i] - 1);


    free(host_JR);
	host_JR = t_JR; // switch
	
	printf("Done.\n");
	printf("JR: "); for (register int i = 0; i < 10; i++) printf("%6.0d", host_JR[i]); printf("\n");
	printf("JC: "); for (register int i = 0; i < 10; i++) printf("%6.0d", host_JC[i]); printf("\n");
	printf("AA: "); for (register int i = 0; i < 10; i++) printf("%6.0lf", host_AA[i]); printf("\n");

#endif

    {
        handle		            = NULL;
        buffer		            = NULL;
        buffer_size             = 0;

		float average			= 0;
        float elapsed           = 0;
        hipEvent_t start, stop;

#ifdef CUSPARSE
		printf("Test: CUSPARSE\n");
#else
		printf("Test: KERNEL\n");
#endif
        // ---- Step 7. Define variables
		const float alpha	    = 1;
		const float beta	    = 0;

		float host_y[N]		    = {0, };
        float host_x[M];
        for (auto& elem : host_x) elem = 1; // Set x to have all ones.

		float* device_x		    = NULL;
        float* device_y		    = NULL;
        
        hipsparseSpMatDescr_t sp_mtx; // device
        hipsparseDnVecDescr_t dn_x, dn_y; // device
        
        CUSPARSE_ERR(hipsparseCreate(&handle));

        CUDA_ERR(hipMalloc((void**)&device_x, sizeof(float) * M));
		CUDA_ERR(hipMalloc((void**)&device_y, sizeof(float) * N));

		CUDA_ERR(hipMemcpy(device_x, host_x, sizeof(float) * M, hipMemcpyHostToDevice));
        CUDA_ERR(hipMemcpy(device_y, host_y, sizeof(float) * N, hipMemcpyHostToDevice));
#ifdef CSR
        CUDA_ERR(hipMalloc((void**)&device_JR, sizeof(int) * (M + 1)));
		CUDA_ERR(hipMalloc((void**)&device_JC, sizeof(int) * NZ));

		CUDA_ERR(hipMemcpy(device_JR, host_JR, sizeof(int) * (M + 1), hipMemcpyHostToDevice));
		CUDA_ERR(hipMemcpy(device_JC, host_JC, sizeof(int) * NZ, hipMemcpyHostToDevice));
#endif

#ifdef CUSPARSE
#ifndef CSR // when COO
        CUSPARSE_ERR(hipsparseCreateCoo(&sp_mtx, 
                M, N, NZ, device_JR, device_JC, device_AA_sorted,
                HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_R_32F));
#endif
#ifdef CSR
        CUSPARSE_ERR(hipsparseCreateCsr(&sp_mtx,
                M, N, NZ, device_JR, device_JC, device_AA_sorted,
                HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_R_32F));
#endif

        CUSPARSE_ERR(hipsparseCreateDnVec(&dn_x, N, device_x, HIP_R_32F));
        CUSPARSE_ERR(hipsparseCreateDnVec(&dn_y, M, device_y, HIP_R_32F));

#ifndef CSR // when COO
		CUSPARSE_ERR(hipsparseSpMV_bufferSize(
			handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&alpha, sp_mtx, dn_x, &beta, dn_y, HIP_R_32F,
			HIPSPARSE_COOMV_ALG, &buffer_size));
#else	// when CSR
		CUSPARSE_ERR(hipsparseSpMV_bufferSize(
			handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&alpha, sp_mtx, dn_x, &beta, dn_y, HIP_R_32F,
			HIPSPARSE_CSRMV_ALG1, &buffer_size));
#endif
		CUDA_ERR(hipMalloc(&buffer, buffer_size));
#endif

		printf("Iteration start.\n");
        for (register int i = 0; i < test_iterations; i++) {
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start); // Timer start

            
		// ---- Step 9. Do SpMV ----
#ifdef CUSPARSE
#ifndef CSR
			CUSPARSE_ERR(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				&alpha, sp_mtx, dn_x, &beta, dn_y, HIP_R_32F,
				HIPSPARSE_COOMV_ALG, buffer));
#else
			CUSPARSE_ERR(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				&alpha, sp_mtx, dn_x, &beta, dn_y, HIP_R_32F,
				HIPSPARSE_CSRMV_ALG1, buffer));
#endif
#else // Kernel function implementation

#ifdef SCALAR_KERNEL
			// find the minimum block
			int block_num = 1;
			int thread_num = M;

			if (M > 1024) {
				while (block_num * 1024 < M) block_num++;
				thread_num = 1024;
			}
			
			ker_csr_spmv_scalar<<<block_num, thread_num>>>(device_JR, device_JC, device_AA_sorted, device_x, device_y);
#endif
#ifdef VECTOR_KERNEL

			// find the minimum block
			int block_num = 1;
			int thread_num = M * 32;

			if (M > 1024) {
				while (block_num * 1024 < 32 * M) block_num++;
				thread_num = 1024;
			}
			
			ker_csr_spmv_vector<<<block_num, thread_num>>>(device_JR, device_JC, device_AA_sorted, device_x, device_y);

#endif
#endif
            // Record
            hipEventRecord(stop); // timer end
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed, start, stop);
			
			if (i != 0) printf("   Iter %3d, Cached, Elapsed: %fms\n", i + 1, elapsed);
			else printf("   Iter %3d, Elapsed: %fms\n", i + 1, elapsed);
			
			average += elapsed;
			elapsed = 0;
		}		

		printf("Iteration end.\n");
		printf("   Average elapsed time: %lf\n", average / test_iterations);

		// ---- Step 10. Fetch the result ----
		CUDA_ERR(hipMemcpy(host_y, device_y, N * sizeof(float), hipMemcpyDeviceToHost));
		printf("Host memory check...\nhost_y: "); for (int i = 0; i < 10; i++) printf("%9.1f", host_y[i]); printf("\n");

		// ---- Step 11. Destroy ----
		CUSPARSE_ERR(hipsparseDestroySpMat(sp_mtx));
		CUSPARSE_ERR(hipsparseDestroyDnVec(dn_x));
        CUSPARSE_ERR(hipsparseDestroyDnVec(dn_y));
        
        // ---- Step 12. Return resources ----
		if (device_JR) 			hipFree(device_JR);
		if (device_JC) 			hipFree(device_JC);
		if (device_AA_sorted) 	hipFree(device_AA_sorted);
		if (device_x) 			hipFree(device_x);
		if (device_y) 			hipFree(device_y);
		if (buffer) 			hipFree(buffer);
		if (handle) 			hipsparseDestroy(handle);

        hipEventDestroy(start);
		hipEventDestroy(stop);

		
    }

    free(host_JR);
	free(host_JC);
	free(host_AA);

    if (hipDeviceReset() != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

    return 0;
}
