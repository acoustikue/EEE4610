#include "hip/hip_runtime.h"
// Week 4
// cuSPARSE vs Kernel Testbench.
// acoustikue@yonsei.ac.kr
// written by SukJoon Oh

#include <hip/hip_runtime.h>
// #include <>
#include <hipsparse.h>	// hipsparseSpMV

#include <stdio.h>
#include <stdlib.h>

#include "mmio.h"

// Option settings

// #define COO
#define CSR

#define CUSPARSE
// #define SCALAR_KERNEL
// #define VECTOR_KERNEL

#define CUDA_ERR(func)                                                         \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CUSPARSE_ERR(func)                                                     \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}


// Author: SukJoon Oh
// acoustikue@yonsei.ac.kr
// Reads MM file.
void read_matrix(int* argJR, int* argJC, float* argAA, int* argP, char* filename, int* argM, int* argN, int* argNZ) {

	int m = 0;
	int n = 0;
	int nz = 0;

	FILE* MTX;
	MTX = fopen(filename, "r");
	 
	MM_typecode matrix_code;
	
	printf("Reading %s : \n", filename);

	// Read banner, type, etc essential infos
	// Verification steps are ignored.
	if (mm_read_banner(MTX, &matrix_code) != 0) exit(1);
	mm_read_mtx_crd_size(MTX, &m, &n, &nz); // Over max 1025

	*argM = m;
	*argN = n;
	*argNZ = nz;

	argJR	    = (int*)malloc(nz * sizeof(int));
	argJC	    = (int*)malloc(nz * sizeof(int));
    argAA	    = (float*)malloc(nz * sizeof(float));
	argP	    = (int*)malloc(nz * sizeof(int));

	// COO format
	for (register int i = 0; i < nz; i++)
		fscanf(MTX, "%d %d %f\n", &argJR[i], &argJC[i], &argAA[i]);

	fclose(MTX);
	
	printf("(File info)\tm : %d, n : %d, nz : %d\n", m, n, nz);
	printf("Printing samples...");
	for (register int i = 0; i < 10; i++) printf("%4.0d", argJR[i]); printf("\n");
	for (register int i = 0; i < 10; i++) printf("%4.0d", argJC[i]); printf("\n");
	for (register int i = 0; i < 10; i++) printf("%4.0d", argAA[i]); printf("\n");
}


// 
// CSR scalar kernel function
__global__ void ker_csr_spmv_scalar(
	const int* argJR, const int* argJC, const float* argAA,
	const float* arg_x, float* arg_y) {

	int idx		= blockDim.x * blockIdx.x + threadIdx.x;
	float sum	= 0;

	for (int i = argJR[idx] - 1; i < argJR[idx + 1] - 1; i++)
		sum		+= (argAA[i] * arg_x[argJC[i] - 1]);

	arg_y[idx]	+= sum;
};


//
// CSR vector kernel function
__global__ void ker_csr_spmv_vector(
	const int* argJR, const int* argJC, const float* argAA,
	const float* arg_x, float* arg_y) {

	// Thread : 32 * M

	int tid		= blockDim.x * blockIdx.x + threadIdx.x;
	int wid		= tid / 32;
	int lidx	= tid & 31;
	float sum	= 0;

	for (int i = argJR[wid] - 1 + lidx; i < argJR[wid + 1] - 1; i += 32)
		sum += argAA[i] * arg_x[argJC[i] - 1];

	for (int i = 16; i > 0; i /= 2)
		sum += __shfl_down_sync(0xFFFFFFFF, sum, i);

	if (lidx == 0) arg_y[wid] = sum;
};

// ---- main() ----
// Entry
int main(int argc, char* argv[])
{
    int test_iterations = 0;
	int N = 0;
	int M = 0;
	int NZ = 0;

    if (argc == 1 || argc == 2) { printf("Too few arguments.\nProgram exit.\n"); exit(0); }
    if (argc >= 4) { printf("Too many argmuments.\nProgram exit.\n"); exit(0); }

    test_iterations = atoi(argv[1]);
    printf("Target iterations: %d\n", test_iterations);
	printf("File name: %s\n", argv[2]);

	// ---- Step 1. Load info ----
	int* host_JR			= NULL;
	int* host_JC			= NULL;
    float* host_AA			= NULL;
	int* host_P				= NULL;

    read_matrix(host_JR, host_JC, host_AA, host_P, argv[2], &M, &N, &NZ); // prepare elements

	printf("File successfully loaded.\n");

	// ---- Step 2. Handle create, bind a stream ---- 
	int* device_JR			= NULL;
	int* device_JC			= NULL;
	float* device_AA		= NULL;
	float* device_AA_sorted	= NULL;
	int* device_P			= NULL;

	void* buffer			= NULL;
	size_t buffer_size		= 0;

	printf("Preparing for cusparseXcoosort...\n");

	hipsparseHandle_t handle = NULL;
	hipStream_t stream		= NULL;

	CUDA_ERR(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	CUSPARSE_ERR(hipsparseCreate(&handle));
	CUSPARSE_ERR(hipsparseSetStream(handle, stream));

	// ---- Step 3. Allocate Buffer ---- 
	CUSPARSE_ERR(hipsparseXcoosort_bufferSizeExt(handle, M, N, NZ, device_JR, device_JC, &buffer_size));
	printf("Buffer allocation for hipsparseXcoosortByRow : %ld Byte\n", buffer_size);

	CUDA_ERR(hipMalloc((void**)&device_JR, sizeof(int) * NZ));
	CUDA_ERR(hipMalloc((void**)&device_JC, sizeof(int) * NZ));
	CUDA_ERR(hipMalloc((void**)&device_P, sizeof(int) * NZ));
	CUDA_ERR(hipMalloc((void**)&device_AA, sizeof(float) * NZ));
	CUDA_ERR(hipMalloc((void**)&device_AA_sorted, sizeof(float) * NZ));
	CUDA_ERR(hipMalloc((void**)&buffer, sizeof(char) * buffer_size));

	CUDA_ERR(hipMemcpy(device_JR, host_JR, sizeof(int) * NZ, hipMemcpyHostToDevice));
	CUDA_ERR(hipMemcpy(device_JC, host_JC, sizeof(int) * NZ, hipMemcpyHostToDevice));
	CUDA_ERR(hipMemcpy(device_AA, host_AA, sizeof(float) * NZ, hipMemcpyHostToDevice));
	CUDA_ERR(hipDeviceSynchronize());

	printf("Allocation/Memcopy to GPU done.");

	// ---- Step 4. Setup permutation vector P to Identity ---- 
	CUSPARSE_ERR(hipsparseCreateIdentityPermutation(handle, NZ, device_P));

	// ---- Step 5. Sort ---- 
	CUSPARSE_ERR(hipsparseXcoosortByRow(handle, M, N, NZ, device_JR, device_JC, device_P, buffer));
	printf("hipsparseXcoosortByRow done.\n");

	// Gather
	CUSPARSE_ERR(hipsparseSgthr(handle, NZ, device_AA, device_AA_sorted, device_P, HIPSPARSE_INDEX_BASE_ZERO));
	CUDA_ERR(hipDeviceSynchronize());

	// Fetch back
	CUDA_ERR(hipMemcpy(host_JR, device_JR, sizeof(int) * NZ, hipMemcpyDeviceToHost));
	CUDA_ERR(hipMemcpy(host_JC, device_JC, sizeof(int) * NZ, hipMemcpyDeviceToHost));
	CUDA_ERR(hipMemcpy(host_P, device_P, sizeof(int) * NZ, hipMemcpyDeviceToHost));
	CUDA_ERR(hipMemcpy(host_AA, device_AA_sorted, sizeof(float) * NZ, hipMemcpyDeviceToHost));
    CUDA_ERR(hipDeviceSynchronize());
    
    // Free memories
    if (device_P) hipFree(device_P);
	if (device_AA) hipFree(device_AA);
	if (buffer) hipFree(buffer);
	if (handle) hipsparseDestroy(handle);
	if (stream) hipStreamDestroy(stream);

	free(host_P); // Unnecessary

#ifdef CSR
	printf("Converting COO to CSR...\n");

	if (device_JR) hipFree(device_JR);
	if (device_JC) hipFree(device_JC);

    int* t_JR	    = (int*)calloc((M + 1), sizeof(int));

    for (int i = 0; i < M + 1; i++) t_JR[i]++; 
    for (int i = 0; i < NZ; i++) t_JR[host_JR[i]]++;

    free(host_JR);
    host_JR = t_JR; // switch
#endif

    {
        handle		            = NULL;
        buffer		            = NULL;
        buffer_size             = 0;

        float elapsed           = 0;
        hipEvent_t start, stop;

#ifdef CUSPARSE
        printf("Test: CUSPARSE");
        // ---- Step 7. Define variables
		const float alpha	    = 1;
		const float beta	    = 0;

		float host_y[N]		    = {0, };
        float host_x[M];
        for (auto& elem : host_x) elem = 1; // Set x to have all ones.

		float* device_x		    = NULL;
        float* device_y		    = NULL;
        
        hipsparseSpMatDescr_t sp_mtx; // device
        hipsparseDnVecDescr_t dn_x, dn_y; // device
        
        CUSPARSE_ERR(hipsparseCreate(&handle));

        CUDA_ERR(hipMalloc((void**)&device_x, sizeof(float) * M));
		CUDA_ERR(hipMalloc((void**)&device_y, sizeof(float) * N));

		CUDA_ERR(hipMemcpy(device_x, host_x, sizeof(float) * M, hipMemcpyHostToDevice));
        CUDA_ERR(hipMemcpy(device_y, host_y, sizeof(float) * N, hipMemcpyHostToDevice));
#ifdef CSR
        CUDA_ERR(hipMalloc((void**)&device_JR, sizeof(int) * (M + 1)));
		CUDA_ERR(hipMalloc((void**)&device_JC, sizeof(int) * NZ));

		CUDA_ERR(hipMemcpy(device_JR, host_JR, sizeof(int) * (M + 1), hipMemcpyHostToDevice));
		CUDA_ERR(hipMemcpy(device_JC, host_JC, sizeof(int) * NZ, hipMemcpyHostToDevice));
#endif
#ifndef CSR
        CUSPARSE_ERR(hipsparseCreateCoo(&sp_mtx, 
                M, N, NZ, device_JR, device_JC, device_AA_sorted,
                HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_R_32F));
#endif
#ifdef CSR
        CUSPARSE_ERR(hipsparseCreateCsr(&sp_mtx,
                M, N, NZ, device_JR, device_JC, device_AA_sorted,
                HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_R_32F));
#endif

        CUSPARSE_ERR(hipsparseCreateDnVec(&dn_x, N, device_x, HIP_R_32F));
        CUSPARSE_ERR(hipsparseCreateDnVec(&dn_y, M, device_y, HIP_R_32F));

#ifndef CSR
		CUSPARSE_ERR(hipsparseSpMV_bufferSize(
			handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&alpha, sp_mtx, dn_x, &beta, dn_y, HIP_R_32F,
			HIPSPARSE_COOMV_ALG, &buffer_size));
#else
		CUSPARSE_ERR(hipsparseSpMV_bufferSize(
			handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&alpha, sp_mtx, dn_x, &beta, dn_y, HIP_R_32F,
			HIPSPARSE_CSRMV_ALG1, &buffer_size));
#endif
        CUDA_ERR(hipMalloc(&buffer, buffer_size));


        for (register int i = 0; i < test_iterations; i++) {
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start); // Timer start

            
		// ---- Step 9. Do SpMV ----
#ifndef CSR
            CUSPARSE_ERR(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                &alpha, sp_mtx, dn_x, &beta, dn_y, HIP_R_32F,
                HIPSPARSE_COOMV_ALG, buffer));
#else
            CUSPARSE_ERR(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                &alpha, sp_mtx, dn_x, &beta, dn_y, HIP_R_32F,
                HIPSPARSE_CSRMV_ALG1, buffer));
            
            // Record
            hipEventRecord(stop); // timer end
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed, start, stop);

            printf("Iteration %3d, Elapsed: %fms\n", elapsed);
            elapsed = 0;

        }

		// ---- Step 11. Destroy ----
		CUSPARSE_ERR(hipsparseDestroySpMat(sp_mtx));
		CUSPARSE_ERR(hipsparseDestroyDnVec(dn_x));
        CUSPARSE_ERR(hipsparseDestroyDnVec(dn_y));

        // ---- Step 10. Fetch the result ----
        CUDA_ERR(hipMemcpy(host_y, device_y, N * sizeof(float), hipMemcpyDeviceToHost));

        for (int i = 0; i < 10; i++) 
            printf("%9.1f", host_y[i]); // Check
        
        // ---- Step 12. Return resources ----
		if (device_JR) hipFree(device_JR);
		if (device_JC) hipFree(device_JC);
		if (device_AA_sorted) hipFree(device_AA_sorted);
		if (device_x) hipFree(device_x);
		if (device_y) hipFree(device_y);
		if (buffer) hipFree(buffer);
		if (handle) hipsparseDestroy(handle);

        hipEventDestroy(start);
		hipEventDestroy(stop);
#endif
#endif
#ifndef CUSPARSE

#endif
    }





    free(host_JR);
	free(host_JC);
	free(host_AA);

    if (hipDeviceReset() != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

    return 0;
}
